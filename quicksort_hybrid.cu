#include "hip/hip_runtime.h"
// hybrid quicksort (16 milliseconds for 4M elements inside RTX4070)
// when chunk size is greater than 1024, it does quicksort steps
// continues splitting chunks
// when chunk size is 1024 or less, executes parallel odd-even sort

#ifndef __HIPCC__
#define __HIPCC__
#endif
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <>
#include <cuda_device_runtime_api.h>
#include <hip/device_functions.h>

#include<iostream>
#include<vector>

__global__ void quickSortWithoutStreamCompaction(
    unsigned int* arr, unsigned int* leftMem, unsigned int* rightMem, int depth, unsigned int* numTasks,
    int* tasks, int* tasks2);

__global__ void resetNumTasks(unsigned int* arr, unsigned int* leftMem, unsigned int* rightMem, int depth, unsigned int* numTasks,
    int* tasks, int* tasks2)
{
    const int n = numTasks[0];

    if (n > 0)
    {
        numTasks[0] = 0;
        //printf("\n %i \n", n);

        quickSortWithoutStreamCompaction << <n, 1024, 0, cudaStreamTailLaunch >> > (arr, leftMem, rightMem, depth, numTasks, tasks, tasks2);
    }

}

__global__ void copyTasksBack(unsigned int* arr, unsigned int* leftMem, unsigned int* rightMem, int depth, unsigned int* numTasks,
    int* tasks, int* tasks2)
{
    const int id = threadIdx.x;
    const int n = numTasks[0];
    const int steps = 1 + n / 1024;

    for (int i = 0; i < steps; i++)
    {
        const int curId = id + i * 1024;
        if (curId < n)
        {
            tasks[curId * 2] = tasks2[curId * 2];
            tasks[curId * 2 + 1] = tasks2[curId * 2 + 1];
        }
    }
    if (id == 0)
    {
        resetNumTasks << <1, 1, 0, cudaStreamTailLaunch >> > (arr, leftMem, rightMem, depth, numTasks, tasks, tasks2);
    }
}

// task pattern: 
//              task 0      task 1      task 2      task 3      ---> array chunks to sort (no overlap)
//              start stop  start stop  start stop  start stop  ---> tasks buffer
//              block 0     block 1     block 2     block 3     ---> cuda blocks
__global__ void quickSortWithoutStreamCompaction(
    unsigned int* arr, unsigned int* leftMem, unsigned int* rightMem, int depth, unsigned int* numTasks,
    int* tasks, int* tasks2)
{
    const int gr = gridDim.x;

    // 1 block = 1 chunk of data
    const int gid = blockIdx.x;
    const int id = threadIdx.x;

    if (id == 0 && gid == 0)
        copyTasksBack << <1, 1024, 0, cudaStreamTailLaunch >> > (arr, leftMem, rightMem, depth, numTasks, tasks, tasks2);

    const int startIncluded = tasks[gid * 2];
    const int stopIncluded = tasks[gid * 2 + 1];
    const int num = stopIncluded - startIncluded + 1;


    if (num < 2)
        return;

    if (num == 2)
    {
        if (id == 0)
        {
            if (arr[startIncluded] > arr[startIncluded + 1])
            {
                unsigned int tmp = arr[startIncluded];
                arr[startIncluded] = arr[startIncluded + 1];
                arr[startIncluded + 1] = tmp;
            }
        }

        return;
    }


    const int bd = blockDim.x;


    int left = 0;
    int right = 0;
    unsigned int pivot = arr[stopIncluded];

    // if chunk size is 1024 or less, do brute-force sorting
    __shared__ unsigned int cache[1024];

    if (num <= 1024)
    {
        if (id < num)
        {
            cache[id] = arr[startIncluded + id];
        }
    }
    __syncthreads();
    if (num <= 1024)
    {
        for (int i = 0; i < num; i++)
        {
            if (id + 1 < num && (id % 2 == 0))
                if (cache[id + 1] < cache[id])
                {
                    unsigned int tmp = cache[id + 1];
                    cache[id + 1] = cache[id];
                    cache[id] = tmp;
                }
            __syncthreads();
            if (id + 1 < num && !(id % 2 == 0))
                if (cache[id + 1] < cache[id])
                {
                    unsigned int tmp = cache[id + 1];
                    cache[id + 1] = cache[id];
                    cache[id] = tmp;
                }
            __syncthreads();
        }
    }

    if (num <= 1024)
    {
        if (id < num)
        {
            arr[startIncluded + id] = cache[id];
        }
    }
    if (num <= 1024)
        return;

    __shared__ int indexLeft;
    __shared__ int indexRight;
    int indexLeftR = 0;
    int indexRightR = 0;
    if (id == 0)
    {
        indexLeft = 0;
        indexRight = 0;
    }
    __syncthreads();
    {
        const int steps = (num / bd) + 1;
        for (int i = 0; i < steps; i++)
        {
            const int curId = i * bd + id;
            if (curId < num)
            {
                const auto data = arr[curId + startIncluded];
                if (data < pivot)
                    leftMem[startIncluded + atomicAdd(&indexLeft, 1)] = data;
                else
                {
                    if (curId + startIncluded != stopIncluded)
                    {
                        rightMem[startIncluded + atomicAdd(&indexRight, 1)] = data;
                    }
                }
            }
        }
    }
    __syncthreads();
    indexLeftR = indexLeft;
    indexRightR = indexRight;
    if (indexLeftR > 0)
    {
        const int steps = (indexLeftR / bd) + 1;
        for (int i = 0; i < steps; i++)
        {
            const int curId = i * bd + id;
            if (curId < indexLeftR)
            {
                arr[curId + startIncluded] = leftMem[startIncluded + curId];
            }
        }
    }
    if (id == 0)
    {
        arr[startIncluded + indexLeftR] = pivot;
    }
    if (indexRightR > 0)
    {
        const int steps = (indexRightR / bd) + 1;
        for (int i = 0; i < steps; i++)
        {
            const int curId = i * bd + id;
            if (curId + indexLeftR + startIncluded + 1 <= stopIncluded)
            {
                arr[curId + indexLeftR + startIncluded + 1] = rightMem[startIncluded + curId];
            }
        }
    }
    __syncthreads();
    auto nLeft = indexLeftR;
    auto nRight = indexRightR;

    if (id == 0)
    {
        if (nLeft > 1)
        {
            if (startIncluded + nLeft - 1 > startIncluded)
            {
                const int index = atomicAdd(&numTasks[0], 1);
                tasks2[index * 2] = startIncluded;
                tasks2[index * 2 + 1] = startIncluded + nLeft - 1;

            }
        }


        if (nRight > 1)
        {
            if (stopIncluded > startIncluded + nLeft + 1)
            {
                const int index = atomicAdd(&numTasks[0], 1);
                tasks2[index * 2] = startIncluded + nLeft + 1;
                tasks2[index * 2 + 1] = stopIncluded;

            }
        }
    }


}



__global__ void qSortMain(
    unsigned int* arr, unsigned int* leftMem, unsigned int* rightMem, int depth, unsigned int* numTasks,
    int* tasks, int* tasks2)
{

    quickSortWithoutStreamCompaction << <1, 1024 >> > (arr, leftMem, rightMem, depth, numTasks, tasks, tasks2);
}

void test()
{
    constexpr int n = 1024 * 1024 * 4;
    unsigned int* data, * left, * right, * numTasks;
    int* tasks, * tasks2;
    std::vector<unsigned int> hostData(n);
    std::vector<int> hostTasks(2);
    hipMalloc(&data, n * sizeof(unsigned int));
    hipMalloc(&left, n * sizeof(unsigned int));
    hipMalloc(&right, n * sizeof(unsigned int));
    hipMalloc(&numTasks, 2 * sizeof(unsigned int));
    hipMalloc(&tasks, n * sizeof(int));
    hipMalloc(&tasks2, n * sizeof(int));
    unsigned int numTasksHost[2];
    for (int j = 0; j < 5; j++)
    {
        for (int i = 0; i < n; i++)
        {
            hostData[i] = rand();
        }
        numTasksHost[0] = 1; // launch 1 block first
        numTasksHost[1] = 0;
        hostTasks[0] = 0;
        hostTasks[1] = n - 1; // first block's chunk limits: 0 - n-1
        hipMemcpy((void*)data, hostData.data(), n * sizeof(unsigned int), hipMemcpyHostToDevice);
        hipMemcpy((void*)numTasks, numTasksHost, 2 * sizeof(unsigned int), hipMemcpyHostToDevice);
        hipMemcpy((void*)tasks, hostTasks.data(), 2 * sizeof(int), hipMemcpyHostToDevice); // host only gives 1 task with 2 parameters
        qSortMain << <1, 1 >> > (data, left, right, 0, numTasks, tasks, tasks2);
        hipDeviceSynchronize();
        hipMemcpy(hostData.data(), (void*)data, n * sizeof(unsigned int), hipMemcpyDeviceToHost);
        hipMemcpy(numTasksHost, (void*)numTasks, 2 * sizeof(unsigned int), hipMemcpyDeviceToHost);

    }

    bool err = false;
    for (int i = 0; i < n - 1; i++)
        if (hostData[i] > hostData[i + 1])
        {
            std::cout << "error at: " << i << ": " << hostData[i] << std::endl;
            err = true;
            break;
        }
    if (!err)
    {
        std::cout << "quicksort completed successfully" << std::endl;
    }
    hipFree(data);
    hipFree(left);
    hipFree(right);
    hipFree(tasks);
    hipFree(tasks2);
    hipFree(numTasks);
}

int main()
{
    test();

    return 0;
}
