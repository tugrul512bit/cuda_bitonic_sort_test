#ifndef __CUDACC__
#define __CUDACC__
#endif
#include <hip/hip_runtime.h>


#include <cuda_device_runtime_api.h>
#include <hip/device_functions.h>
#include <mma.h>

__global__ void matrixMul(
    half*  a, half*  b, half*  c
    )
{
    const int index = (threadIdx.x + blockIdx.x * blockDim.x);
    hipStream_t stream0;
    hipStreamCreateWithFlags(&stream0, hipStreamNonBlocking);
    constexpr int WMMA_M = 16;
    constexpr int WMMA_N = 16;
    constexpr int WMMA_K = 16;
    constexpr int lda = 16;
    constexpr int ldb = 16;
    constexpr int ldc = 16;

    nvcuda::wmma::fragment<nvcuda::wmma::matrix_a, WMMA_M, WMMA_N, WMMA_K, half, nvcuda::wmma::col_major> a_frag;
    nvcuda::wmma::fragment<nvcuda::wmma::matrix_b, WMMA_M, WMMA_N, WMMA_K, half, nvcuda::wmma::col_major> b_frag;
    nvcuda::wmma::fragment<nvcuda::wmma::accumulator, WMMA_M, WMMA_N, WMMA_K, half> acc_frag;
    nvcuda::wmma::fragment<nvcuda::wmma::accumulator, WMMA_M, WMMA_N, WMMA_K, half> c_frag;

    // all warp threads need to execute this
    nvcuda::wmma::fill_fragment(acc_frag, 0.0f);
    nvcuda::wmma::load_matrix_sync(a_frag, a, lda);
    nvcuda::wmma::load_matrix_sync(b_frag, b, ldb);
    nvcuda::wmma::mma_sync(acc_frag, a_frag, b_frag, acc_frag);
    nvcuda::wmma::load_matrix_sync(c_frag, c, ldc, nvcuda::wmma::mem_col_major);
    
    // all warp threads need to execute this
    for (int i = 0; i < c_frag.num_elements; i++)    
        c_frag.x[i] += acc_frag.x[i];
    
    nvcuda::wmma::store_matrix_sync(c, c_frag, ldc, nvcuda::wmma::mem_col_major);
}

#include<iostream>
void test2()
{
    const int n = 16*16;
    half *dvcA,*dvcB,*dvcC;
    hipMalloc(&dvcA, n * sizeof(half));
    hipMalloc(&dvcB, n * sizeof(half));
    hipMalloc(&dvcC, n * sizeof(half));

    half *hstA,*hstB,*hstC;
    hipHostAlloc(&hstA, n * sizeof(half), hipHostMallocDefault);
    hipHostAlloc(&hstB, n * sizeof(half), hipHostMallocDefault);
    hipHostAlloc(&hstC, n * sizeof(half), hipHostMallocDefault);
    
 
    for (int i = 0; i < n; i++)
    {
        hstA[i] = i / 100.0f; // division to not overflow 16-bit floats
        hstB[i] = i / 100.0f;
        hstC[i] = 0;
    }
    std::cout << "tensor" << std::endl;

    hipStream_t stream0;
    hipStreamCreate(&stream0);

    hipEvent_t evt,evt2;
    hipEventCreate(&evt);
    hipEventCreate(&evt2);
    hipEventRecord(evt, stream0);
    
    hipDeviceProp_t props;
    hipGetDeviceProperties(&props, 0);
    for (int i = 0; i < 1; i++)
    {
        hipMemcpyAsync(dvcA, hstA, n * sizeof(half), ::hipMemcpyHostToDevice, stream0);
        hipMemcpyAsync(dvcB, hstB, n * sizeof(half), ::hipMemcpyHostToDevice, stream0);
        hipMemcpyAsync(dvcC, hstC, n * sizeof(half), ::hipMemcpyHostToDevice, stream0);

        // launch single warp (may not be 32 in future)
        matrixMul<<<1, props.warpSize,0, stream0>>>(dvcA,dvcB,dvcC);
        

        hipMemcpyAsync(hstC, dvcC, n * sizeof(half), ::hipMemcpyDeviceToHost, stream0);
    }
    
    hipEventRecord(evt2, stream0);
    hipEventSynchronize(evt2);
    float tim;
    hipEventElapsedTime(&tim, evt, evt2);


    std::cout<<(float)hstC[5+3*16]<< std::endl;
    float acc = 0.0f;
    for (int i = 0; i < 16; i++)
    {
        acc += (i+3*16) * (i * 16+5);
    }
    std::cout << acc << std::endl;

    hipHostFree(hstA);
    hipFree(dvcA);
    
}

int main()
{
    test2();
	return 0;
}
